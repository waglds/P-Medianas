#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>

#include <thrust\host_vector.h>
#include <thrust\device_vector.h>
#include <thrust\sort.h>
#include <thrust\extrema.h>

#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <limits.h>
#include <string>
#include <sstream>
#include <book.h>

using namespace std;

// Numero de vertices do grafo.
#define V 65
// Numero de medianas do problema.
#define MEDIANAS 5
// Numero de possiveis solucoes.
#define Q_SOLUCOES 8259888

#define FLAG 0

static int matrizSolucoes[Q_SOLUCOES * MEDIANAS];

static int valorMedianaSol[Q_SOLUCOES];

static int matrizGrafo[V][V];

static int matrizDistancias[V * V];

__host__ int minDistance(const int dist[], const bool sptSet[]){

	int min = INT_MAX, min_index;

	for (int v = 0; v < V; v++){

		if (sptSet[v] == false && dist[v] <= min){

			min = dist[v], min_index = v;

		}

	}

	return min_index;

}

__host__ int dijkstra(int graph[V][V], int src, int dst){

	int distancia[V];

	bool sptSet[V];

	for (int i = 0; i < V; i++){
    
		distancia[i] = INT_MAX, sptSet[i] = false;
	
	}

	distancia[src] = 0;

	for (int count = 0; count < V-1; count++){

		int u = minDistance(distancia, sptSet);

		sptSet[u] = true;

		for (int v = 0; v < V; v++){

			if (!sptSet[v] && graph[u][v] && distancia[u] != INT_MAX
            		&& distancia[u]+graph[u][v] < distancia[v]){

				distancia[v] = distancia[u] + graph[u][v];

			}

		}

	}

	return distancia[dst];

}

__host__ int distanciaNos(int src, int dst){
	
	return matrizDistancias[src * V + dst];

}

__global__ void checar(int medianas, int i, int matrizSolucoes[Q_SOLUCOES * MEDIANAS], int *solucaoAtual){
		
	int tx = threadIdx.x;
	
	if(tx < medianas){
		
		solucaoAtual[tx] = matrizSolucoes[i * medianas + tx];

	}

}


__host__ void pMediana(int graph[V][V]){

	int *solucaoAtual = (int*)malloc(sizeof(int) * MEDIANAS), 
		*hSolucao, *hMatriz;
	int *atual, *menor, *Solucoes;

	hipMalloc((void**)&hSolucao, sizeof(int) * MEDIANAS);
	hipMalloc((void**)&hMatriz, sizeof(int) * Q_SOLUCOES * MEDIANAS);
	hipMemcpy(hMatriz, matrizSolucoes, sizeof(int) * Q_SOLUCOES * MEDIANAS, hipMemcpyHostToDevice);

	for(int i = 0; i < Q_SOLUCOES; i++){
		
		checar<<<1, MEDIANAS>>>(MEDIANAS, i, hMatriz, hSolucao);

		hipMemcpy(solucaoAtual, hSolucao, sizeof(int) * MEDIANAS, hipMemcpyDeviceToHost);

		int medianaAtual = 0;

		for(int k = 0; k < V; k++){

			bool isParteSolucao = false;

			for(int j = 0; j < MEDIANAS; j++){
				
				if(k == solucaoAtual[j]){

					isParteSolucao = true;
					break;

				}

			}

		
			if(!isParteSolucao){

				int distSolucoes[MEDIANAS];

				for (int j = 0; j < MEDIANAS; j++){

					distSolucoes[j] = distanciaNos(solucaoAtual[j], k);

				}

				int menorValor = distSolucoes[0];

				for(int j = 1; j < MEDIANAS; j++){

					if(distSolucoes[j] < menorValor){

						menorValor = distSolucoes[j];

					}

				}

				medianaAtual = medianaAtual + menorValor;

			}

		}

		valorMedianaSol[i] = medianaAtual;

	}

	free(solucaoAtual);
	hipFree(hSolucao);
	hipFree(hMatriz);

}

__host__ void carregarGrafo(){

	std::stringstream numGrafo;
	numGrafo << V;

	std::string str = "grafos/" + numGrafo.str();

	const char* path = str.c_str(); 

	FILE* arquivo = fopen(path, "r");

	if (arquivo == NULL){

        fprintf(stderr, "Arquivo do Grafo nao foi aberto!\n");
        exit(1);

    }

    for (int i = 0; i < V; i++){

        for (int j = 0; j < V; j++){

            int a = 0;
            fscanf(arquivo, "%d", &a);
            matrizGrafo[i][j] = a;

        }

    }

}

__host__ void carregarSolucoes(){

	if(MEDIANAS == 2){

		int num_matr = 0;

		for (int i = 0; i < (V - 1); i++){

			for (int j = (i + 1); j < (V); j++){

				matrizSolucoes[num_matr * MEDIANAS + 0] = i;
				matrizSolucoes[num_matr * MEDIANAS + 1] = j;

				num_matr++;

			}

		 }

	 } else if(MEDIANAS == 3){

		int num_matr = 0;

		for (int i = 0; i < (V - 2); i++){

			for (int j = (i + 1); j < (V - 1); j++){

				for (int k = (j + 1); k < (V); k++){

					matrizSolucoes[num_matr * MEDIANAS + 0] = i;
					matrizSolucoes[num_matr * MEDIANAS + 1] = j;
					matrizSolucoes[num_matr * MEDIANAS + 2] = k;

					num_matr++;

				}

			}

		}

	 } else if(MEDIANAS == 5){

		int num_matr = 0;

		for (int i = 0; i < (V - 4); i++){

			for (int j = (i + 1); j < (V - 3); j++){

				for (int k = (j + 1); k < (V - 2); k++){

					for (int l = (k + 1); l < (V - 1); l++){

						for (int m = (l + 1); m < V; m++){

							matrizSolucoes[num_matr * MEDIANAS + 0] = i;
							matrizSolucoes[num_matr * MEDIANAS + 1] = j;
							matrizSolucoes[num_matr * MEDIANAS + 2] = k;
							matrizSolucoes[num_matr * MEDIANAS + 3] = l;
							matrizSolucoes[num_matr * MEDIANAS + 4] = m;

							num_matr++;

						}

					}

				}

			}

		}

	}else{

		cout << "Esse valor de mediana nao eh usado!" << endl;

	}

}

int main(){

	cout << "Carregando o Problema...\n" << endl;
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start);

	carregarGrafo();
    carregarSolucoes();
	
	
	for(int i = 0; i < V; i++){
		
		for(int j = 0; j < V; j++){
			
			matrizDistancias[i * V + j] = dijkstra(matrizGrafo, i, j);

		}

	}

    cout << "Resolvendo...\n" << endl;

	pMediana(matrizGrafo);
		
	thrust::device_ptr<int> dp = thrust::device_pointer_cast(valorMedianaSol);
	
	int *pos = thrust::min_element(valorMedianaSol, valorMedianaSol + Q_SOLUCOES);

	unsigned int indexMelhor = thrust::distance(dp, (thrust::device_ptr<int>)pos);

	hipEventRecord(stop);
	hipEventSynchronize(stop);	
	
	for(int i = 0; i < MEDIANAS; i++){
	
		cout << matrizSolucoes[indexMelhor * MEDIANAS + i] << " ";

	}

	cout  << endl;

	float milliseconds = 0;

	hipEventElapsedTime(&milliseconds, start, stop);

	cout << "Processing time: " << milliseconds / 1000.0 << " s" << endl;

	cout << endl;
	
	hipEventDestroy(start);
	hipEventDestroy(stop);

    return 0;

}
